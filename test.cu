#include "hip/hip_runtime.h"
#include "perf_timer.h"
#include "test.h"
#include "thread_pool.h"
#include "utils.h"

#include <atomic>
#include <cstdio>
#include <hip/hip_runtime.h>
#include <math.h>

namespace pk
{

const unsigned int TOTAL_ELEMENTS = ( 1 << 20 );
const unsigned int BLOCK_SIZE = TOTAL_ELEMENTS / 4;
const unsigned int NUM_BLOCKS = TOTAL_ELEMENTS / BLOCK_SIZE;

static void add( int n, float* x, float* y )
{
    for ( int i = 0; i < n; i++ )
        y[ i ] = x[ i ] + y[ i ];
}


void testCPU()
{
    printf( "Single thread: %d elements\n", TOTAL_ELEMENTS );

    PerfTimer t;

    float* x = new float[ TOTAL_ELEMENTS ];
    float* y = new float[ TOTAL_ELEMENTS ];

    // initialize x and y arrays on the host
    for ( int i = 0; i < TOTAL_ELEMENTS; i++ ) {
        x[ i ] = 1.0f;
        y[ i ] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    add( TOTAL_ELEMENTS, &x[ 0 ], &y[ 0 ] );

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for ( int i = 0; i < TOTAL_ELEMENTS; i++ )
        maxError = fmax( maxError, fabs( y[ i ] - 3.0f ) );

    printf( "Max error: %f\n", maxError );

    // Free memory
    delete[] x;
    delete[] y;

    printf( "Elapsed ms: %d\n\n", (uint32_t)t.ElapsedMilliseconds() );
}


typedef struct {
    int                start;
    int                num;
    float*             x;
    float*             y;
    int                total;
    std::atomic<bool>* complete;
} thread_context_t;


static bool _cpu_add_thread( void* context, uint32_t tid )
{
    thread_context_t* ctx = (thread_context_t*)context;

    add( ctx->num, ctx->x, ctx->y );

    if ( ctx->start + ctx->num >= ctx->total ) {
        *( ctx->complete ) = true;
    }

    return true;
}


void testCPUThreaded()
{
    printf( "Threaded: %d blocks of %d\n", NUM_BLOCKS, BLOCK_SIZE );

    thread_pool_t     tp       = threadPoolInit( 4 );
    thread_context_t* contexts = new thread_context_t[ NUM_BLOCKS ];

    PerfTimer t;

    float* x = new float[ TOTAL_ELEMENTS ];
    float* y = new float[ TOTAL_ELEMENTS ];

    // initialize x and y arrays on the host
    for ( int i = 0; i < TOTAL_ELEMENTS; i++ ) {
        x[ i ] = 1.0f;
        y[ i ] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    std::atomic<bool> complete = false;
    for ( int i = 0; i < NUM_BLOCKS; i++ ) {
        thread_context_t& ctx = contexts[ i ];
        ctx.start             = i * BLOCK_SIZE;
        ctx.num               = BLOCK_SIZE;
        ctx.total             = TOTAL_ELEMENTS;
        ctx.x                 = &x[ ctx.start ];
        ctx.y                 = &y[ ctx.start ];
        ctx.complete          = &complete;

        threadPoolSubmitJob( tp, _cpu_add_thread, &ctx, THREAD_POOL_SUBMIT_BLOCKING );
    }

    // Wait for threads to complete
    while ( !complete ) {
        //delay( 1 );
    }

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for ( int i = 0; i < TOTAL_ELEMENTS; i++ )
        maxError = fmax( maxError, fabs( y[ i ] - 3.0f ) );

    printf( "Max error: %f\n", maxError );

    // Free memory
    delete[] x;
    delete[] y;

    printf( "Elapsed ms: %d\n\n", (uint32_t)t.ElapsedMilliseconds() );
    threadPoolDeinit( tp );
}


__global__ static void addCUDA( int n, float* x, float* y )
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;

    for (int i = index; i < n; i += stride) {
        y[ i ] = x[ i ] + y[ i ];
    }
}


void testCUDA()
{
    int numDevices = 0;
    int device = 0;
    CHECK_CUDA(hipGetDeviceCount( &numDevices ));
    printf("%d CUDA devices found.\n", numDevices);

    int numSMs;
    CHECK_CUDA(hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, device));
    printf("%d SMs on device %d\n", numSMs, device);

    int blockSize = 256;
    int numBlocks = (TOTAL_ELEMENTS + blockSize - 1) / blockSize;

    printf( "CUDA: %d blocks of %d\n", numBlocks, blockSize );

    PerfTimer t;

    float* x = nullptr;
    float* y = nullptr;
    CHECK_CUDA(hipMallocManaged( &x, TOTAL_ELEMENTS * sizeof( float ) ));
    CHECK_CUDA(hipMallocManaged( &y, TOTAL_ELEMENTS * sizeof( float ) ));

    // initialize x and y arrays on the host
    for ( int i = 0; i < TOTAL_ELEMENTS; i++ ) {
        x[ i ] = 1.0f;
        y[ i ] = 2.0f;
    }

    // Run kernel on 1M elements on the CPU
    addCUDA<<<numBlocks, blockSize>>>( TOTAL_ELEMENTS, x, y );

    // Wait for threads to complete
    CHECK_CUDA(hipDeviceSynchronize());

    // Check for errors (all values should be 3.0f)
    float maxError = 0.0f;
    for ( int i = 0; i < TOTAL_ELEMENTS; i++ )
        maxError = fmax( maxError, fabs( y[ i ] - 3.0f ) );

    printf( "Max error: %f\n", maxError );

    // Free memory
    CHECK_CUDA(hipFree( x ));
    CHECK_CUDA(hipFree( y ));

    printf( "Elapsed ms: %d\n\n", (uint32_t)t.ElapsedMilliseconds() );
}

} // namespace pk
